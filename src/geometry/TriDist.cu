#include "hip/hip_runtime.h"
/*************************************************************************\

  Copyright 1999 The University of North Carolina at Chapel Hill.
  All Rights Reserved.

  Permission to use, copy, modify and distribute this software and its
  documentation for educational, research and non-profit purposes, without
  fee, and without a written agreement is hereby granted, provided that the
  above copyright notice and the following three paragraphs appear in all
  copies.

  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL BE
  LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL, INCIDENTAL, OR
  CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS, ARISING OUT OF THE
  USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE UNIVERSITY
  OF NORTH CAROLINA HAVE BEEN ADVISED OF THE POSSIBILITY OF SUCH
  DAMAGES.

  THE UNIVERSITY OF NORTH CAROLINA SPECIFICALLY DISCLAIM ANY
  WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
  MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.  THE SOFTWARE
  PROVIDED HEREUNDER IS ON AN "AS IS" BASIS, AND THE UNIVERSITY OF
  NORTH CAROLINA HAS NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT,
  UPDATES, ENHANCEMENTS, OR MODIFICATIONS.

  The authors may be contacted via:

  US Mail:             E. Larsen
                       Department of Computer Science
                       Sitterson Hall, CB #3175
                       University of N. Carolina
                       Chapel Hill, NC 27599-3175

  Phone:               (919)962-1749

  EMail:               geom@cs.unc.edu


\**************************************************************************/

//--------------------------------------------------------------------------
// File:   TriDist.cpp
// Author: Eric Larsen
// Description:
// contains SegPoints() for finding closest points on a pair of line
// segments and TriDist() for finding closest points on a pair of triangles
//--------------------------------------------------------------------------

#include "TriDist.h"
#include <pthread.h>

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

namespace hispeed{

inline void __cudaSafeCall( hipError_t err)
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed: %s\n",
                 hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

// copy
__device__
inline void
VcV(float Vr[3], const float V[3])
{
  Vr[0] = V[0];  Vr[1] = V[1];  Vr[2] = V[2];
}

// minus
__device__
inline void
VmV(float Vr[3], const float V1[3], const float V2[3])
{
  Vr[0] = V1[0] - V2[0];
  Vr[1] = V1[1] - V2[1];
  Vr[2] = V1[2] - V2[2];
}

// plus
__device__
inline void
VpV(float Vr[3], const float V1[3], const float V2[3])
{
  Vr[0] = V1[0] + V2[0];
  Vr[1] = V1[1] + V2[1];
  Vr[2] = V1[2] + V2[2];
}

// plus after product
__device__
inline void
VpVxS(float Vr[3], const float V1[3], const float V2[3], float s)
{
  Vr[0] = V1[0] + V2[0] * s;
  Vr[1] = V1[1] + V2[1] * s;
  Vr[2] = V1[2] + V2[2] * s;
}

// dot product
__device__
inline float
VdotV(const float V1[3], const float V2[3])
{
  return (V1[0]*V2[0] + V1[1]*V2[1] + V1[2]*V2[2]);
}

__device__
inline float SegDist_kernel(const float *S, const float *T,
							const float *A, const float *B)
{

	float t = 0.0, u = 0.0, dist = 0.0, t1 = 0.0;
	float ST[3]; // temporary vector S->T
	VmV(ST,T,S);
	float A_dot_A = VdotV(A,A);
	float B_dot_B = VdotV(B,B);
	if(A_dot_A==0||B_dot_B==0){
		return DBL_MAX;
	}
	float A_dot_B = VdotV(A,B);
	float A_dot_ST = VdotV(A,ST);
	float B_dot_ST = VdotV(B,ST);

	// t parameterizes ray P,A
	// u parameterizes ray Q,B

	// compute t for the closest point on ray P,A to
	// ray Q,B

	float denom = A_dot_A*B_dot_B - A_dot_B*A_dot_B;
	if(denom == 0){
		t = 0;
	}else{
		t = (A_dot_ST*B_dot_B - B_dot_ST*A_dot_B) / denom;
	}

	// find u for point on ray Q,B closest to point at t
	// B_dot_B can never be 0
	u = (t*A_dot_B - B_dot_ST)/B_dot_B;
	// if u is on segment Q,B, t and u correspond to
	// closest points, otherwise, recompute and
	// clamp t

	if (u <= 0) {
		u = 0;
	} else if (u >= 1) {
		u = 1;
	}

	t = (A_dot_B*u+A_dot_ST)/A_dot_A;

	if(t<=0){
		t = 0;
	} else if(t >= 1){
		t = 1;
	}

	t1 = A[0]*t-ST[0]-B[0]*u;
	dist += t1*t1;
	t1 = A[1]*t-ST[1]-B[1]*u;
	dist += t1*t1;
	t1 = A[2]*t-ST[2]-B[2]*u;
	dist += t1*t1;

	return dist;
}


//	float idf = (float)id;
//	float AA[3]={1,1,1},BB[3]={1,1,1},
//			SS[6]={2+idf,2+idf,2+idf,3+idf,3+idf,3+idf},
//			TT[6]={0+idf,0+idf,0+idf,1+idf,1+idf,1+idf};
//	float dd = SegDist_kernel(SS, TT, AA, BB);


__global__
void SegDist_cuda(const float *S, const float *T,
				  const float *A, const float *B,
				  float *dist){
	int x = blockIdx.x;
	int y = threadIdx.x;
	int id = x*blockDim.x+y;
	const float *cur_S = S+x*6;
	const float *cur_T = T+y*6;
	const float *cur_A = A+x*3;
	const float *cur_B = B+y*3;
	float dd = SegDist_kernel(cur_S, cur_T, cur_A, cur_B);

	if(dist[id]>dd){
		dist[id] = dd;
	}
}


__global__
void SegDist_cuda_new(const float *S, const float *T,
				  	  const float *A, const float *B,
				  	  float *dist, int voxel_size){
	// computing which segment in set1 of voxel with
	// voxel_id
	int voxel_id = blockIdx.x;
	int segment_id = threadIdx.x;
	// update the pointers for current thread
	const float *cur_S = S+6*voxel_id*400+6*segment_id;
	const float *cur_T = T+6*voxel_id*400;
	const float *cur_A = A+3*voxel_id*400+3*segment_id;
	const float *cur_B = B+3*voxel_id*400;

	float min_dist = DBL_MAX;
	// go over all the segments in set2
	for(int i = 0;i<voxel_size;i++){
		float dd = SegDist_kernel(cur_S, cur_T, cur_A, cur_B);
		if(min_dist>dd){
			min_dist = dd;
		}
		cur_T += 6;
		cur_B += 3;
	}
	// initialize the minimum distance
	if(segment_id == 0){
		dist[voxel_id] = min_dist;
	}
	// update the minimum distance
	for(int i=1;i<voxel_size;i++){
		if(i==segment_id){
			if(dist[voxel_id]>min_dist){
				dist[voxel_id] = min_dist;
			}
		}
	}
}

__global__
void get_max(float *d, float *max_d, int batch)
{
	int id = threadIdx.x;
	float min_val = DBL_MAX;
	float *cur_d = d+batch*id;
	for(int i=0;i<batch;i++){
		if(cur_d[i]<min_val){
			min_val = cur_d[i];
		}
	}
	max_d[id] = min_val;
}

__global__
void get_vector_kernel(float *S, float *A){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	float *cur_A = A+id*3;
	VmV(cur_A, S+id*6+3, S+id*6);
}


// the computing capacity of the gpu
int max_len_x = 200;
int max_len_y = 512;

float SegDist_batch_gpu(const float *S, const float *T, int size1, int size2){

	struct timeval start = get_cur_time();
	float *d_S, *d_T, *d_A, *d_B;
	float *d_dist;

	int len_x = min(max_len_x, size1);
	int len_y = min(max_len_y, size2);
	float min_dist = DBL_MAX;

	// copy data into device
	hipMalloc(&d_S, size1*6*sizeof(float));
	hipMalloc(&d_T, size2*6*sizeof(float));
	hipMemcpy(d_S, S, size1*6*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_T, T, size2*6*sizeof(float), hipMemcpyHostToDevice);

	// some temporary space for computation
	hipMalloc(&d_A, size1*3*sizeof(float));
	hipMalloc(&d_B, size2*3*sizeof(float));

	// compute the vectors of segments in S and T, save to A and B
	for(int i=0;i<size1;){
		float *cur_S = d_S+i*6;
		float *cur_A = d_A+i*3;

		int b_num;
		int t_num;
		if(size1>i+max_len_x*max_len_y){
			b_num = max_len_x;
			t_num = max_len_y;
			i += max_len_x*max_len_y;
		}else if(size1 < i+max_len_y){
			b_num = 1;
			t_num = size1-i;
			i = size1;
		}else{
			b_num = (size1-i)/max_len_y;
			t_num = max_len_y;
			i += b_num*max_len_y;
		}
		get_vector_kernel<<<b_num, t_num>>>(cur_S, cur_A);
	}
	for(int i=0;i<size2;){
		float *cur_T = d_T+i*6;
		float *cur_B = d_B+i*3;

		int b_num;
		int t_num;
		if(size2>i+max_len_x*max_len_y){
			b_num = max_len_x;
			t_num = max_len_y;
			i += max_len_x*max_len_y;
		}else if(size2 < i+max_len_y){
			b_num = 1;
			t_num = size2-i;
			i = size2;
		}else{
			b_num = (size2-i)/max_len_y;
			t_num = max_len_y;
			i += b_num*max_len_y;
		}
		get_vector_kernel<<<b_num, t_num>>>(cur_T, cur_B);
	}
	hipDeviceSynchronize();

	// space for the distances got from gpu
	hipMalloc(&d_dist, len_x*len_y*sizeof(float));
	float *dist = new float[len_x*len_y];
	for(int i=0;i<len_x*len_y;i++){
		dist[i] = DBL_MAX;
	}
	hipMemcpy(d_dist, dist, len_x*len_y*sizeof(float), hipMemcpyHostToDevice);
	cout<<"preprocessing data takes "<<get_time_elapsed(start)<<" ms"<<endl;

	start = get_cur_time();
	const float *cur_S, *cur_T, *cur_A, *cur_B;
	int times = 0;
	for(int i=0;i<size1;i+=len_x){
		for(int j=0;j<size2;j+=len_y){
			times++;
			cur_S = d_S+i*6;
			cur_T = d_T+j*6;
			cur_A = d_A+i*3;
			cur_B = d_B+j*3;
			int tsize1 = min(len_x, size1-i);
			int tsize2 = min(len_y, size2-j);
			SegDist_cuda<<<tsize1, tsize2>>>(cur_S, cur_T, cur_A, cur_B, d_dist);
		}
	}
	hipDeviceSynchronize();
	double time_elapsed = get_time_elapsed(start);

	cout<<"run "<<times<<" rounds in "<<time_elapsed<<" ms, each round takes "<<time_elapsed/times<<" ms "<<endl;

	start = get_cur_time();
	hipMemcpy(dist, d_dist, len_x*len_y*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<len_x*len_y;i++){
		if(min_dist > dist[i]){
			min_dist = dist[i];
		}
	}
	min_dist = sqrt(min_dist);
	cout<<"reduce minimum distance takes "<<get_time_elapsed(start)<<" ms"<<endl;

	start = get_cur_time();
	hipFree(d_S);
	hipFree(d_T);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_dist);
	delete dist;
	cout<<"clean spaces takes "<<get_time_elapsed(start)<<" ms"<<endl;


	return min_dist;
}


void SegDist_gpu_fixed_batch(const float *S, const float *T, int batch_size, int batch_num, float *result){

	struct timeval start = get_cur_time();
	float *d_S, *d_T, *d_A, *d_B, *d_dist;

	size_t segment_num = batch_size*batch_num;

	// segment data in device
	hipMalloc(&d_S, segment_num*6*sizeof(float));
	hipMalloc(&d_T, segment_num*6*sizeof(float));
	// space for the results in GPU
	hipMalloc(&d_dist, batch_num*sizeof(float));
	// some temporary space for computation
	hipMalloc(&d_A, segment_num*3*sizeof(float));
	hipMalloc(&d_B, segment_num*3*sizeof(float));
	cout<<"allocating space in takes "<<get_time_elapsed(start, true)<<" ms"<<endl;

	hipMemcpy(d_S, S, segment_num*6*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_T, T, segment_num*6*sizeof(float), hipMemcpyHostToDevice);
	cout<<"copying data in takes "<<get_time_elapsed(start, true)<<" ms"<<endl;

	// compute the vectors of segments in S and T, save to A and B
	get_vector_kernel<<<batch_num, batch_size>>>(d_S, d_A);
	get_vector_kernel<<<batch_num, batch_size>>>(d_T, d_B);

	hipDeviceSynchronize();
	cout<<"preprocessing data takes "<<get_time_elapsed(start, true)<<" ms"<<endl;

	// compute the distance in parallel
	SegDist_cuda_new<<<batch_num, batch_size>>>(d_S, d_T, d_A, d_B, d_dist, batch_size);
	hipDeviceSynchronize();
	cout<<"distances computations takes "<<get_time_elapsed(start, true)<<" ms"<< endl;

	hipMemcpy(result, d_dist, batch_num*sizeof(float), hipMemcpyDeviceToHost);
	cout<<"copying result out takes "<<get_time_elapsed(start, true)<<" ms"<<endl;

	hipFree(d_S);
	hipFree(d_T);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_dist);
	cout<<"clean spaces takes "<<get_time_elapsed(start)<<" ms"<<endl;
}

}
