#include "hip/hip_runtime.h"
/*************************************************************************\

  Copyright 1999 The University of North Carolina at Chapel Hill.
  All Rights Reserved.

  Permission to use, copy, modify and distribute this software and its
  documentation for educational, research and non-profit purposes, without
  fee, and without a written agreement is hereby granted, provided that the
  above copyright notice and the following three paragraphs appear in all
  copies.

  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL BE
  LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL, INCIDENTAL, OR
  CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS, ARISING OUT OF THE
  USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE UNIVERSITY
  OF NORTH CAROLINA HAVE BEEN ADVISED OF THE POSSIBILITY OF SUCH
  DAMAGES.

  THE UNIVERSITY OF NORTH CAROLINA SPECIFICALLY DISCLAIM ANY
  WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
  MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.  THE SOFTWARE
  PROVIDED HEREUNDER IS ON AN "AS IS" BASIS, AND THE UNIVERSITY OF
  NORTH CAROLINA HAS NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT,
  UPDATES, ENHANCEMENTS, OR MODIFICATIONS.

  The authors may be contacted via:

  US Mail:             E. Larsen
                       Department of Computer Science
                       Sitterson Hall, CB #3175
                       University of N. Carolina
                       Chapel Hill, NC 27599-3175

  Phone:               (919)962-1749

  EMail:               geom@cs.unc.edu


\**************************************************************************/

//--------------------------------------------------------------------------
// File:   TriDist.cpp
// Author: Eric Larsen
// Description:
// contains SegPoints() for finding closest points on a pair of line
// segments and TriDist() for finding closest points on a pair of triangles
//--------------------------------------------------------------------------

#include "geometry.h"

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

#define CUDA_EXECUTE(call) \
	do{\
	call; \
	hipError_t err = hipGetLastError();\
	if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));\
	}while(0);\

#define CUDA_SAFE_CALL(call) \
	do {\
		hipError_t err = call;\
		if (hipSuccess != err) {\
			fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
					__FILE__, __LINE__, hipGetErrorString(err) );\
			exit(EXIT_FAILURE);\
		}\
	} while (0);

namespace hispeed{

inline void __cudaSafeCall( hipError_t err)
{
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed: %s\n",
                 hipGetErrorString( err ) );
        exit( -1 );
    }
    return;
}

inline void check_execution(){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
}

// copy
__device__
inline void
VcV_d(float Vr[3], const float V[3])
{
  Vr[0] = V[0];  Vr[1] = V[1];  Vr[2] = V[2];
}

// minus
__device__
inline void
VmV_d(float Vr[3], const float V1[3], const float V2[3])
{
  Vr[0] = V1[0] - V2[0];
  Vr[1] = V1[1] - V2[1];
  Vr[2] = V1[2] - V2[2];
}

// plus
__device__
inline void
VpV_d(float Vr[3], const float V1[3], const float V2[3])
{
  Vr[0] = V1[0] + V2[0];
  Vr[1] = V1[1] + V2[1];
  Vr[2] = V1[2] + V2[2];
}

// plus after product
__device__
inline void
VpVxS_d(float Vr[3], const float V1[3], const float V2[3], float s)
{
  Vr[0] = V1[0] + V2[0] * s;
  Vr[1] = V1[1] + V2[1] * s;
  Vr[2] = V1[2] + V2[2] * s;
}

// dot product
__device__
inline float
VdotV_d(const float V1[3], const float V2[3])
{
  return (V1[0]*V2[0] + V1[1]*V2[1] + V1[2]*V2[2]);
}

// return the distance of two segments
__device__
inline float SegDist_kernel(const float *S, const float *T,
							const float *A, const float *B)
{

	float t = 0.0, u = 0.0, dist = 0.0, t1 = 0.0;
	float ST[3]; // temporary vector S->T
	VmV_d(ST,T,S);
	float A_dot_A = VdotV_d(A,A);
	float B_dot_B = VdotV_d(B,B);
	if(A_dot_A==0||B_dot_B==0){
		return DBL_MAX;
	}
	float A_dot_B = VdotV_d(A,B);
	float A_dot_ST = VdotV_d(A,ST);
	float B_dot_ST = VdotV_d(B,ST);

	// t parameterizes ray P,A
	// u parameterizes ray Q,B

	// compute t for the closest point on ray P,A to
	// ray Q,B

	float denom = A_dot_A*B_dot_B - A_dot_B*A_dot_B;
	if(denom == 0){
		t = 0;
	}else{
		t = (A_dot_ST*B_dot_B - B_dot_ST*A_dot_B) / denom;
	}

	// find u for point on ray Q,B closest to point at t
	// B_dot_B can never be 0
	u = (t*A_dot_B - B_dot_ST)/B_dot_B;
	// if u is on segment Q,B, t and u correspond to
	// closest points, otherwise, recompute and
	// clamp t

	if (u <= 0) {
		u = 0;
	} else if (u >= 1) {
		u = 1;
	}

	t = (A_dot_B*u+A_dot_ST)/A_dot_A;

	if(t<=0){
		t = 0;
	} else if(t >= 1){
		t = 1;
	}

	t1 = A[0]*t-ST[0]-B[0]*u;
	dist += t1*t1;
	t1 = A[1]*t-ST[1]-B[1]*u;
	dist += t1*t1;
	t1 = A[2]*t-ST[2]-B[2]*u;
	dist += t1*t1;

	return dist;
}


//	float idf = (float)id;
//	float AA[3]={1,1,1},BB[3]={1,1,1},
//			SS[6]={2+idf,2+idf,2+idf,3+idf,3+idf,3+idf},
//			TT[6]={0+idf,0+idf,0+idf,1+idf,1+idf,1+idf};
//	float dd = SegDist_kernel(SS, TT, AA, BB);

//
//__global__
//void SegDist_cuda(const float *data, const long *offset_size,
//				  	  const float *vec, float *dist){
//	// computing which segment in set1 of voxel with
//	// which batch
//	int batch_id = blockIdx.x;
//	// which segment in the batch
//	int segment_id = threadIdx.x;
//	long offset = offset_size[batch_id*3];
//	long size1 = offset_size[batch_id*3+1];
//	// the segment_size is the maximum size of the set 1
//	// thus some may does not have an segment
//	if(segment_id>=size1){
//		return;
//	}
//	long size2 = offset_size[batch_id*3+2];
//
//	// update the pointers for current thread
//	const float *cur_S = data+6*(offset+segment_id);
//	const float *cur_T = data+6*(offset+size1);
//	const float *cur_A = vec+3*(offset+segment_id);
//	const float *cur_B = vec+3*(offset+size1);
//
//	float min_dist = DBL_MAX;
//	// go over all the segments in set2
//	for(int i = 0;i<size2;i++){
//		float dd = SegDist_kernel(cur_S, cur_T, cur_A, cur_B);
//		if(min_dist>dd){
//			min_dist = dd;
//		}
//		cur_T += 6;
//		cur_B += 3;
//	}
//
//	// initialize the minimum distance
//	if(segment_id == 0){
//		dist[voxel_id] = min_dist;
//	}
//
//	// update the minimum distance
//	for(int i=1;i<size1;i++){
//		if(i==segment_id){
//			if(dist[voxel_id]>min_dist){
//				dist[voxel_id] = min_dist;
//			}
//		}
//	}
//}

__global__
void SegDist_cuda(const float *data, const uint *offset_size,
				  const float *vec, float *dist, uint batch_num){
	// which batch
	int batch_id = blockIdx.x*blockDim.x+threadIdx.x;
	if(batch_id>=batch_num){
		return;
	}
	uint offset1 = offset_size[batch_id*4];
	uint size1 = offset_size[batch_id*4+1];
	uint offset2 = offset_size[batch_id*4+2];
	uint size2 = offset_size[batch_id*4+3];
	// update the pointers for current thread
	const float *cur_S = data+6*offset1;
	const float *cur_A = vec+3*offset1;

	float min_dist = DBL_MAX;
	// go over all the segment pairs
	for(int i=0;i<size1;i++){
		const float *cur_T = data+6*offset2;
		const float *cur_B = vec+3*offset2;
		for(int j=0;j<size2;j++){
			float dd = SegDist_kernel(cur_S, cur_T, cur_A, cur_B);
			if(min_dist>dd){
				min_dist = dd;
			}
			cur_T += 6;
			cur_B += 3;
		}
		cur_S += 6;
		cur_A += 3;
	}
	dist[batch_id] = min_dist;
}

__global__
void get_max(float *d, float *max_d, int batch)
{
	int id = threadIdx.x;
	float min_val = DBL_MAX;
	float *cur_d = d+batch*id;
	for(int i=0;i<batch;i++){
		if(cur_d[i]<min_val){
			min_val = cur_d[i];
		}
	}
	max_d[id] = min_val;
}

__global__
void get_vector_kernel(float *data, float *vec, int segment_num){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<segment_num){
		VmV_d(vec+id*3, data+id*6+3, data+id*6);
	}
}
//
//if(false){
//	int cur_iter = 0;
//	do{
//		if(cur_iter%num_per_iter==0||(cur_iter+num_per_iter)>=batch_num){
//			int step = ((cur_iter+num_per_iter)>=batch_num)?(cur_iter+num_per_iter-batch_num):num_per_iter;
//			long cur_offset = offset_size[cur_iter*3];
//			float *cur_data = d_data+cur_offset*6;
//			float *cur_vec = d_vec+cur_offset*3;
//			long *cur_os = d_os+cur_iter*3;
//			float *cur_dist = d_dist+cur_iter;
//			//SegDist_cuda<<<step, batch_size>>>(cur_data, cur_os, cur_vec, cur_dist);
//			check_execution();
//		}
//		cur_iter++;
//	}while(cur_iter<batch_num);
//}else{
//	SegDist_cuda<<<batch_num/1024+1, 1024>>>(d_data, d_os, d_vec, d_dist, batch_num);
//	check_execution();
//}

char *d_cuda = NULL;
// by default 1GB
size_t cuda_mem_size = (1<<30)/3*4;
void init_cuda(){
	if(d_cuda==NULL){
		struct timeval start = get_cur_time();
		CUDA_SAFE_CALL(hipMalloc(&d_cuda, cuda_mem_size));
		report_time("allocating space in GPU", start);
	}
}

void clean_cuda(){
	if(d_cuda){
		struct timeval start = get_cur_time();
		CUDA_SAFE_CALL(hipFree(d_cuda));
		report_time("clean space in GPU", start);
		d_cuda = NULL;
	}
}

/*
 * data: contains the segments of the meshes mentioned in this join.
 * offset_size:  contains the offset in the data for each batch, and the sizes of two data sets
 * result: for the returned results for each batch
 * batch_num: number of computed batches
 *
 * */
void SegDist_batch_gpu(const float *data, const uint *offset_size, float *result, const uint batch_num, const uint segment_num){

	// initialize cuda memory if not done yet
	init_cuda();

	struct timeval start = get_cur_time();
	// allocate memory in GPU
	char *cur_d_cuda = d_cuda;
	// segment data in device
	float *d_data = (float *)(cur_d_cuda);
	cur_d_cuda += 6*sizeof(float)*segment_num;
	// some temporary space for computation
	float *d_vec = (float *)(cur_d_cuda);
	cur_d_cuda += 3*sizeof(float)*segment_num;
	// space for the results in GPU
	float *d_dist = (float *)(cur_d_cuda);
	cur_d_cuda += sizeof(float)*batch_num;
	// space for the offset and size information in GPU
	uint *d_os = (uint *)(cur_d_cuda);

	CUDA_SAFE_CALL(hipMemcpy(d_data, data, segment_num*6*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_os, offset_size, batch_num*4*sizeof(uint), hipMemcpyHostToDevice));
	//report_time("copying data to GPU", start);

	// compute the vectors of segments in data, save to d_vec
	get_vector_kernel<<<segment_num/1024+1,1024>>>(d_data, d_vec, segment_num);
	check_execution();
	// compute the distance in parallel
	SegDist_cuda<<<batch_num/1024+1, 1024>>>(d_data, d_os, d_vec, d_dist, batch_num);
	check_execution();
	hipDeviceSynchronize();
	//report_time("distances computations", start);

	CUDA_SAFE_CALL(hipMemcpy(result, d_dist, batch_num*sizeof(float), hipMemcpyDeviceToHost));
	//report_time("copy data out", start);

}

}
