/*
 *
 * with some common gpu related operations
 * */

#include <hip/hip_runtime.h>
#include "cuda_util.h"
#include "mygpu.h"

using namespace std;


namespace hispeed{


vector<gpu_info *> get_gpus(){
	vector<gpu_info *> gpus;
	int num_gpus = 0;
	hipGetDeviceCount(&num_gpus);
	for (int i = 0; i < num_gpus; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		gpu_info *info = new gpu_info();
		info->busy = false;
		info->mem_size = prop.totalGlobalMem/1024/1024*4/5;
		info->device_id = i;
		// we allocate 2G mem for each gpu
		if(info->mem_size>2048){
			info->mem_size = 2048;
		}
		gpus.push_back(info);
	}
	return gpus;
}

void print_gpus(){
	int num_gpus = 0;
	hipGetDeviceCount(&num_gpus);
	for (int i = 0; i < num_gpus; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("  Memory size (MB): %ld\n\n", prop.totalGlobalMem/1024/1024);
	}
}


void init_gpu(gpu_info *gpu){
	assert(gpu);
	if(!gpu->d_data){
		struct timeval start = get_cur_time();
		CUDA_SAFE_CALL(hipSetDevice(gpu->device_id));
		CUDA_SAFE_CALL(hipMalloc((void **)&gpu->d_data, gpu->mem_size*1024*1024));
		assert(gpu->d_data);
		cerr<<gpu->mem_size<<" MB memory size is allocated for GPU "<<gpu->device_id<<endl;
		report_time("allocating space in GPU", start);
	}
}

void clean_gpu(gpu_info *gpu){
	if(gpu->d_data){
		struct timeval start = get_cur_time();
		hipSetDevice(gpu->device_id);
		CUDA_SAFE_CALL(hipFree(gpu->d_data));
		report_time("clean space in GPU", start);
		gpu->d_data = NULL;
	}
}

void initialize(){
	hipInit(0);
}

}
